
#include "hip/hip_runtime.h"
#include ""

#include <hip/device_functions.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

hipCtx_t      hContext = 0;

#define CUDA_CHECK( fn ) do { \
		hipError_t status = (fn); \
		if ( hipSuccess != status ) { \
			const char* errstr; \
			hipDrvGetErrorString(status, &errstr); \
			printf("CUDA Driver Failure (line %d of file %s):\n\t%s returned 0x%x (%s)\n", __LINE__, __FILE__, #fn, status, errstr); \
			exit(EXIT_FAILURE); \
						} \
		} while (0)


void gflops(const char* ident, int N, float ms, int repeat)
{

	double msecPerMatrixMul = ms / repeat;
	long int Gflops = (128*16*1024)*1024*2.0*1e-9f;//*2 for MUL and ADD
	double gigaFlops = (Gflops) / (msecPerMatrixMul) * 1000.0f ;
	printf("ms = %f \n", msecPerMatrixMul);
	printf("%s GFLOPS: %.2f (size: %d, iterations: %d)\n", ident, gigaFlops, N, repeat);

}



int main()
{
	//-----------------sample_data_config---------------------
	int NBLOCK = 1024;
	int N = 2048*NBLOCK + 1024*30;//1024032;//1023985;             160768.0*4/1024/1024=0.61328125MB  
	int M = 1024;//16;
	int P = 2048*NBLOCK;
	size_t sizeSampleFloat = N * sizeof(float);
	size_t sizeFilterFloat = M * sizeof(float);//16 * 4;
	size_t sizeResultFloat = P * sizeof(float);

	int repeat = 4;

	dim3 threads(128, 1, 1);
	dim3 grid(NBLOCK, 1, 1);

	hipError_t error;

	char deviceName[32];
	int count, ordinal, major, minor;
	hipDevice_t  hDevice;
	hipEvent_t hStart, hStop;
	hipDeviceptr_t devH, devX, devY;


	// ------Initialize the Driver API and find a device-----
	CUDA_CHECK(hipInit(0));
	CUDA_CHECK(hipGetDeviceCount(&count));
	for (ordinal = 0; ordinal < count; ordinal++)
	{
		CUDA_CHECK(hipDeviceGet(&hDevice, ordinal));
		CUDA_CHECK(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, hDevice));
		CUDA_CHECK(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, hDevice));
		CUDA_CHECK(hipDeviceGetName(deviceName, sizeof(deviceName), hDevice));
		if (major >= 5 && minor >= 2)
		{
			//printf("Using: Id:%d %s (%d.%d)\n\n", ordinal, deviceName, major, minor);
			break;
		}
	}
	if (ordinal == count)
	{
		printf("No compute 5.0 device found, exiting.\n");
		exit(EXIT_FAILURE);
	}


	//-----------------device_test------------------------

	int device = 0;
	error = hipSetDevice(0);

	if (error != hipSuccess)
	{
		printf("device error");
		exit(EXIT_FAILURE);
	}

	else printf("device:  %d  \n", device);

	hipDeviceProp_t deviceProp;
	error = hipGetDeviceProperties(&deviceProp, 0);

	if (error != hipSuccess)
	{
		printf("DeviceProperties error");
		exit(EXIT_FAILURE);
	}

	

	//-----------------------host----------------------------

	float* H = (float*)malloc(sizeFilterFloat);
	float* X = (float*)malloc(sizeSampleFloat);
	float* Y = (float*)malloc(sizeResultFloat);
	float* T = (float*)malloc(sizeResultFloat);

	for (int i = 0; i < N ; i++) //
	{
		X[i] = (float)rand()/1000;//(float)1.0;//
		//if(X[i] == (float)16) X[i]=0;
	}

	for (int i = 0; i < M; i++) //
	{
		H[i] = (float)rand()/1000;//(float)i;// (i % 2);//(float)rand();//(float)1.0;//
	}

	for (int i = 0; i < P; i++) //
	{
		Y[i] = (float)0.0;
		T[i] = (float)0.0;
	}

	//conv calculate

	for (int i = 0; i < P; i++) 
	{
		int k = i;
		for (int j = 1024; j > 0; j--)
		{
			T[i] += H[j - 1] * X[k];
			k++;
		}
	}

	

	//-----------------------Dev----------------------------

	CUDA_CHECK(hipCtxCreate(&hContext, 0, hDevice));

	CUDA_CHECK(hipEventCreateWithFlags(&hStart, hipEventBlockingSync)); // hipEventDefault 
	CUDA_CHECK(hipEventCreateWithFlags(&hStop, hipEventBlockingSync));

	CUDA_CHECK(hipMalloc(&devH, sizeFilterFloat));
	CUDA_CHECK(hipMalloc(&devX, sizeSampleFloat));
	CUDA_CHECK(hipMalloc(&devY, sizeResultFloat));

	CUDA_CHECK(hipMemcpyHtoD(devH, H, sizeFilterFloat));
	CUDA_CHECK(hipMemcpyHtoD(devX, X, sizeSampleFloat));


	//---------------------Kernel----------------------------

	printf("Computing result using CUDA Kernel...\n");

	// Load the cubin
	hipModule_t hModule;
	CUDA_CHECK(hipModuleLoad(&hModule, "conv.cubin"));


	// Load the kernel function
	hipFunction_t hKernel;
	CUDA_CHECK(hipModuleGetFunction(&hKernel, hModule, "conv_kernel_128"));

	void * params[] = {&devH, &devX, &devY};

	float totalTime = 0;
	// Launch the kernel repeat times.. but break it up into pieces so as not to lock things up.

	CUDA_CHECK(hipEventCreateWithFlags(&hStart, hipEventBlockingSync)); // hipEventDefault 
	CUDA_CHECK(hipEventCreateWithFlags(&hStop, hipEventBlockingSync));

	while (repeat > 0)
	{
		float ms;
		int r = repeat;
		CUDA_CHECK(hipEventRecord(hStart, NULL));


		for (int i = 0; i < repeat; i++)
			CUDA_CHECK(hipModuleLaunchKernel(hKernel, grid.x, 1, 1, threads.x, 1, 1, 0, 0, params, 0));

		CUDA_CHECK(hipEventRecord(hStop, NULL));
		CUDA_CHECK(hipEventSynchronize(hStop));	
		CUDA_CHECK(hipEventElapsedTime(&ms, hStart, hStop));
		
		totalTime += ms;
		//gflops("conv_kernel_128", P, totalTime, repeat);
		repeat -= r;


	}

	//CUDA_CHECK(hipModuleLaunchKernel(hKernel, grid.x, grid.y, 1, threads.x, 1, 1, 0, 0, params, 0));
	//CUDA_CHECK(hipModuleLaunchKernel(hKernel, grid.x, grid.y, 1, threads.x, 1, 1, 0, 0, params, 0));

	CUDA_CHECK(hipModuleUnload(hModule));


	// Copy result from device to host
	CUDA_CHECK(hipMemcpyDtoH(Y, devY, sizeResultFloat));
	//CUDA_CHECK(hipMemcpyDtoH(H, devH, sizeFilterFloat));
	//CUDA_CHECK(hipMemcpyDtoH(X, devX, sizeSampleFloat));


	//for (int i = 0; i<20; i++) {
		//if (Y[i] != 0.0f)
	//		printf("Y[%d] = %f \n", i, Y[i]);
	//}

	//for (int i = 2048*0; i<2048*780; i++) {
	//	if (Y[i] != T[i])//1024.0f)
	//		printf("Y[%d] = %f \n", i, Y[i]);
	//}

	//-----------------------free----------------------------



	// Cleanup and shutdown of cuda
	CUDA_CHECK(hipFree(devH));
	CUDA_CHECK(hipFree(devX));
	CUDA_CHECK(hipFree(devY));


	for (int i = 0; i<1024*1; i++)
		printf("Y[%d] = %f --- and --- T[%d] = %f    error = %f\n", i, Y[i], i, T[i], T[i] - Y[i]);


	for (int i = 0; i<P; i++) 
	{
		if ( (Y[i]-T[i] > 1) || (Y[i]-T[i] < -1) )
			printf("Y[%d] = %f --- but --- T[%d] = %f    error = %f\n", i, Y[i], i, T[i], T[i] - Y[i]);
	}


//for (int i = 2048*0; i<2048*1; i++) {
	//if (Y[i] != 1024.0f)
//		printf("T[%d] = %f \n", i, T[i]);
//}

	free(H);
	free(X);
	free(Y);
	free(T);

	CUDA_CHECK(hipEventDestroy(hStart));
	CUDA_CHECK(hipEventDestroy(hStop));
	CUDA_CHECK(hipCtxDestroy(hContext));
	hContext = 0;


	hipDeviceReset();
	printf("done\n");


	return EXIT_SUCCESS;


}






