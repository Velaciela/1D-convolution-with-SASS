
#include "hip/hip_runtime.h"
#include ""

#include <hip/device_functions.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

hipCtx_t      hContext = 0;

#define CUDA_CHECK( fn ) do { \
		hipError_t status = (fn); \
		if ( hipSuccess != status ) { \
			const char* errstr; \
			hipDrvGetErrorString(status, &errstr); \
			printf("CUDA Driver Failure (line %d of file %s):\n\t%s returned 0x%x (%s)\n", __LINE__, __FILE__, #fn, status, errstr); \
			exit(EXIT_FAILURE); \
						} \
		} while (0)


void gflops(const char* ident, int N, float ms, int repeat)
{

	float msecPerMatrixMul = ms / repeat;
	double flopsPerMatrixMul = (N+16-1) * 16.0 ;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf("ms = %f \n", msecPerMatrixMul);
	printf("%s GFLOPS: %.2f (size: %d, iterations: %d)\n", ident, gigaFlops, N, repeat);

}



int main()
{
	//-----------------sample_data_config---------------------
	int N = 1024032;//1024032;//1023985;
	int M = 16;//16;
	int P = 1024000;
	size_t sizeSampleFloat = N * 4;
	size_t sizeFilterFloat = M * 4;//16 * 4;
	size_t sizeResultFloat = P * 4;

	dim3 threads(32, 1, 1);
	dim3 grid(2000, 1, 1);

	hipError_t error;

	char deviceName[32];
	int count, ordinal, major, minor;
	hipDevice_t  hDevice;
	hipEvent_t hStart, hStop;
	hipDeviceptr_t devH, devX, devY;


	// ------Initialize the Driver API and find a device-----
	CUDA_CHECK(hipInit(0));
	CUDA_CHECK(hipGetDeviceCount(&count));
	for (ordinal = 0; ordinal < count; ordinal++)
	{
		CUDA_CHECK(hipDeviceGet(&hDevice, ordinal));
		CUDA_CHECK(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, hDevice));
		CUDA_CHECK(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, hDevice));
		CUDA_CHECK(hipDeviceGetName(deviceName, sizeof(deviceName), hDevice));
		if (major >= 5 && minor >= 2)
		{
			//printf("Using: Id:%d %s (%d.%d)\n\n", ordinal, deviceName, major, minor);
			break;
		}
	}
	if (ordinal == count)
	{
		printf("No compute 5.0 device found, exiting.\n");
		exit(EXIT_FAILURE);
	}


	//-----------------device_test------------------------

	int device = 0;
	error = hipSetDevice(0);

	if (error != hipSuccess)
	{
		printf("device error");
		exit(EXIT_FAILURE);
	}

	else printf("device:  %d  \n", device);

	hipDeviceProp_t deviceProp;
	error = hipGetDeviceProperties(&deviceProp, 0);

	if (error != hipSuccess)
	{
		printf("DeviceProperties error");
		exit(EXIT_FAILURE);
	}

	//-----------------------host----------------------------

	float* H = (float*)malloc(sizeFilterFloat);
	float* X = (float*)malloc(sizeSampleFloat);
	float* Y = (float*)malloc(sizeResultFloat);
	float* T = (float*)malloc(sizeResultFloat);

	for (int i = 0; i < N ; i++) 
	{
		X[i] = (float)rand()/1000;
	}

	for (int i = 0; i < M; i++) 
	{
		H[i] = (float)rand()/1000;
	}

	for (int i = 0; i < P; i++) //
	{
		Y[i] = (float)0.0;
		T[i] = (float)0.0;
	}


	for (int i = 0; i < P; i++) 
	{
		int k = i;
		for (int j = 16; j > 0; j--)
		{
			T[i] += H[j - 1] * X[k];
			k++;
		}
	}


	//-----------------------Dev----------------------------

	CUDA_CHECK(hipCtxCreate(&hContext, 0, hDevice));

	CUDA_CHECK(hipEventCreateWithFlags(&hStart, hipEventBlockingSync)); // hipEventDefault 
	CUDA_CHECK(hipEventCreateWithFlags(&hStop, hipEventBlockingSync));

	CUDA_CHECK(hipMalloc(&devH, sizeFilterFloat));
	CUDA_CHECK(hipMalloc(&devX, sizeSampleFloat));
	CUDA_CHECK(hipMalloc(&devY, sizeResultFloat));

	CUDA_CHECK(hipMemcpyHtoD(devH, H, sizeFilterFloat));
	CUDA_CHECK(hipMemcpyHtoD(devX, X, sizeSampleFloat));


	//---------------------Kernel----------------------------

	printf("Computing result using CUDA Kernel...\n");

	// Load the cubin
	hipModule_t hModule;
	CUDA_CHECK(hipModuleLoad(&hModule, "conv.cubin"));

	// Load the kernel function
	hipFunction_t hKernel;
	CUDA_CHECK(hipModuleGetFunction(&hKernel, hModule, "conv_kernel_32"));

	void * params[] = {&devH, &devX, &devY};

	int repeat = 20;
	float totalTime = 0;
	// Launch the kernel repeat times.. but break it up into pieces so as not to lock things up.


	CUDA_CHECK(hipEventCreateWithFlags(&hStart, hipEventBlockingSync)); // hipEventDefault 
	CUDA_CHECK(hipEventCreateWithFlags(&hStop, hipEventBlockingSync));


	while (repeat > 0)
	{
		float ms;
		int r = repeat;
		CUDA_CHECK(hipEventRecord(hStart, NULL));

		for (int i = 0; i < repeat; i++)
			CUDA_CHECK(hipModuleLaunchKernel(hKernel, grid.x, 1, 1, threads.x, 1, 1, 0, 0, params, 0));

		CUDA_CHECK(hipEventRecord(hStop, NULL));
		CUDA_CHECK(hipEventSynchronize(hStop));
		CUDA_CHECK(hipEventElapsedTime(&ms, hStart, hStop));

		totalTime += ms;

		//gflops("conv_kernel_32", N, ms, repeat);

		repeat -= r;


	}

	//CUDA_CHECK(hipModuleLaunchKernel(hKernel, grid.x, grid.y, 1, threads.x, 1, 1, 0, 0, params, 0));
	//CUDA_CHECK(hipModuleLaunchKernel(hKernel, grid.x, grid.y, 1, threads.x, 1, 1, 0, 0, params, 0));

	CUDA_CHECK(hipModuleUnload(hModule));

	printf("first time done\n");



	// Copy result from device to host
	CUDA_CHECK(hipMemcpyDtoH(Y, devY, sizeResultFloat));
	CUDA_CHECK(hipMemcpyDtoH(H, devH, sizeFilterFloat));
	CUDA_CHECK(hipMemcpyDtoH(X, devX, sizeSampleFloat));



	for (int i = 1024*0; i<1024*1; i++)
		printf("Y[%d] = %f --- and --- T[%d] = %f    delta = %f\n", i, Y[i], i, T[i], T[i] - Y[i]);


	for (int i = 1024*0; i<P; i++) 
	{
		if (Y[i] - T[i] > 1e-2)
			printf("Y[%d] = %f --- but --- T[%d] = %f    delta = %f\n", i, Y[i], i, T[i], T[i] - Y[i]);
	}

	//-----------------------free----------------------------



	// Cleanup and shutdown of cuda
	CUDA_CHECK(hipFree(devH));
	CUDA_CHECK(hipFree(devX));
	CUDA_CHECK(hipFree(devY));

	free(H);
	free(X);
	free(Y);

	CUDA_CHECK(hipEventDestroy(hStart));
	CUDA_CHECK(hipEventDestroy(hStop));

	//CUBLAS_CHECK( hipblasDestroy(hCublas) );
	//hCublas  = 0;
	CUDA_CHECK(hipCtxDestroy(hContext));
	hContext = 0;


	hipDeviceReset();

	printf("done\n");


	return EXIT_SUCCESS;


}






